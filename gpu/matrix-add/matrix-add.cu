#include "hip/hip_runtime.h"
#include <stdio.h>
#include <files.h>
#define CHECK_CORRECTNESS

#define N  10000

__global__ void matrixAddGPU( double * a, double * b, double * c )
{

  int row_begin = blockIdx.x * blockDim.x + threadIdx.x;
  int col_begin = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_row = gridDim.x * blockDim.x;
  int stride_col = gridDim.y * blockDim.y;

  for(int row = row_begin; row < N ;row += stride_row) {
        for(int col= col_begin; col< N ; col+= stride_col) {
                c[row * N + col] = a[row*N+col] + b[row*N+col];
        }
  }
}

void matrixAddCPU( double * a, double * b, double * c )
{

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      c[row * N + col] = a[row*N+col]+b[row*N+col];
    }
}

int main()
{
        hipError_t cudaStatus;

  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("SM:%d\n",numberOfSMs);//80

  double *a, *b, *c_gpu;

  unsigned long long size = (unsigned long long)N * N * sizeof (double); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_gpu, size);
  read_values_from_file("matrix_a_data", a, size);
  read_values_from_file("matrix_b_data", b, size);

  //if too large,invalid configuration argument
  dim3 threads_per_block(32,32,1);
  dim3 number_of_blocks (16*numberOfSMs,16*numberOfSMs, 1);
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c_gpu, size, deviceId);
  matrixAddGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
                fprintf(stderr, "call matrixAddGPU error: %s\n", hipGetErrorString(cudaStatus));
                return -1;
        }

  hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding

  // Call the CPU version to check our work
    // Compare the two answers to make sure they are equal
  bool error = false;
  #ifdef CHECK_CORRECTNESS
    double *c_cpu;
    hipMallocManaged (&c_cpu, size);
    matrixAddCPU( a, b, c_cpu );
    for( int row = 0; row < N && !error; ++row )
      for( int col = 0; col < N && !error; ++col )
        if (c_cpu[row * N + col] != c_gpu[row * N + col])
        {
          printf("FOUND ERROR at c[%d][%d]\n", row, col);
          error = true;
          break;
        }
    hipFree( c_cpu );
  #endif
  if (!error)
    printf("Success!\n");
  write_values_to_file("result/matrix_c_data", c_gpu, size);
  // Free all our allocated memory
  hipFree(a);
  hipFree(b);
  hipFree( c_gpu );
}
